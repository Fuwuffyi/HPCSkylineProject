#include "hip/hip_runtime.h"
#if _XOPEN_SOURCE < 600
#define _XOPEN_SOURCE 600
#endif

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

#include "hpc.h"

/**
 * Point data structure.
 */
typedef struct {
   float *P;       /* pointer to flat array of coordinates (size N * D) */
   unsigned int N; /* number of points */
   unsigned int D; /* dimension of all points */
} points_t;

/**
 * Read dimension, number of points, and coordinates from stdin.
 */
void read_input(points_t *points) {
   char buf[1024];
   unsigned int N, D;
   float *P;
   // Read the dimension
   if (1 != scanf("%u", &D)) {
      fprintf(stderr, "FATAL: can not read the dimension\n");
      exit(EXIT_FAILURE);
   }
   assert(D >= 2);
   // Skip rest of line
   if (NULL == fgets(buf, sizeof(buf), stdin)) {
      fprintf(stderr, "FATAL: can not read the first line\n");
      exit(EXIT_FAILURE);
   }
   // Read point count
   if (1 != scanf("%u", &N)) {
      fprintf(stderr, "FATAL: can not read the number of points\n");
      exit(EXIT_FAILURE);
   }
   // Allocate point array
   P = (float *)malloc(D * N * sizeof(*P));
   assert(P);
   // Read all points
   for (unsigned int i = 0; i < N; ++i) {
      for (unsigned int k = 0; k < D; ++k) {
         if (1 != scanf("%f", &(P[i * D + k]))) {
            fprintf(stderr, "FATAL: failed to get coordinate %u of point %u\n", k, i);
            exit(EXIT_FAILURE);
         }
      }
   }
   points->P = P;
   points->N = N;
   points->D = D;
}

/**
 * Frees points memory.
 */
void free_points(points_t *points) {
   free(points->P);
   points->P = NULL;
   points->N = points->D = 0;
}

/**
 * Check if point p dominates point q in all dimensions.
 * Returns 1 if p >= q in every coordinate and p > q in at least one.
 * Returns 0 otherwise.
 */
__device__ char dominates(const float *p, const float *q, const unsigned int D) {
   char strictly_greater = 0;
   for (unsigned int k = 0; k < D; ++k) {
      if (p[k] < q[k]) return 0;
      strictly_greater |= (p[k] > q[k]);
   }
   return strictly_greater;
}

/**
 * Compute the skyline of a set of points.
 * Uses an array of skyline_flags (size N) marking whether each point remains in the skyline (1 = in, 0 = out)
 * Returns the number of skyline points.
 */
__global__ void skyline(const float *points_data, char *skyline_flags, const unsigned int N, const unsigned int D) {
   const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i >= N) return;
   const float *pi = points_data + i * D;
   for (unsigned int j = 0; j < N; ++j) {
      if (!skyline_flags[j]) continue;
      const float *pj = points_data + j * D;
      if (dominates(pi, pj, D)) {
         skyline_flags[j] = 0;
      }
   }
}

/**
 * Output the skyline to stdout in the expected format.
 * First prints D, then r (number of skyline points), then each skyline point.
 */
void print_skyline(const points_t *points, const char *skyline_flags, const unsigned int r) {
   const unsigned int D = points->D;
   const unsigned int N = points->N;
   const float *P = points->P;
   // Print dimension and skyline size
   printf("%u\n", D);
   printf("%u\n", r);
   // Print each skyline point's coordinates
   for (unsigned int i = 0; i < N; ++i) {
      if (!skyline_flags[i]) continue;
      for (unsigned int k = 0; k < D; ++k) {
         printf("%f ", P[i * D + k]);
      }
      printf("\n");
   }
}

int main(int argc, char *argv[]) {
   points_t points;
   if (argc != 1) {
      fprintf(stderr, "Usage: %s < input_file > output_file\n", argv[0]);
      return EXIT_FAILURE;
   }
   // Read in data
   read_input(&points);
   const unsigned int N = points.N;
   const unsigned int D = points.D;
   const size_t point_bytes = (size_t)N * D * sizeof(float);
   const size_t flag_bytes = (size_t)N * sizeof(char);
   // Allocate local flags
   char *h_skyline_flags = (char *)malloc(points.N * sizeof(*h_skyline_flags));
   assert(h_skyline_flags);
   // Allocate data on the gpu
   float *d_P;
   char *d_skyline_flags;
   cudaSafeCall(hipMalloc(&d_P, point_bytes));
   cudaSafeCall(hipMalloc(&d_skyline_flags, flag_bytes));
   cudaSafeCall(hipMemcpy(d_P, points.P, point_bytes, hipMemcpyHostToDevice));
   // Calculate blocks
   const unsigned int threads_per_block = 256;
   const unsigned int blocks = (N + threads_per_block - 1) / threads_per_block;
   // Run the skyline algorithm
   const double tstart = hpc_gettime();
   cudaSafeCall(hipMemset(d_skyline_flags, 1, flag_bytes));
   skyline<<<blocks, threads_per_block>>>(d_P, d_skyline_flags, N, D);
   cudaSafeCall(hipDeviceSynchronize());
   // Copy data from host
   cudaSafeCall(hipMemcpy(h_skyline_flags, d_skyline_flags, flag_bytes, hipMemcpyDeviceToHost));
   // TODO: Implement this within the kernel?
   unsigned int r = 0;
   for (unsigned int i = 0; i < N; ++i) {
      if (h_skyline_flags[i] == 1) {
         ++r;
      }
   }
   const double elapsed = hpc_gettime() - tstart;
   // Print results
   print_skyline(&points, h_skyline_flags, r);
   fprintf(stderr, "\n\t%u points\n", points.N);
   fprintf(stderr, "\t%u dimensions\n", points.D);
   fprintf(stderr, "\t%u points in skyline\n\n", r);
   fprintf(stderr, "Execution time (s) %f\n", elapsed);
   // Free and exit
   free_points(&points);
   cudaSafeCall(hipFree(d_P));
   free(h_skyline_flags);
   cudaSafeCall(hipFree(d_skyline_flags));
   return EXIT_SUCCESS;
}
